#include "hip/hip_runtime.h"
#include <cstdio> 
#include <future>

#define NUMERIC_TYPE float
#define C(x) x
#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr, "%s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

__constant__ int xsz;
__constant__ NUMERIC_TYPE end_time = C(1000.0);
__managed__ bool finished = false;
__managed__ NUMERIC_TYPE t = C(0.0);
__device__ NUMERIC_TYPE next_snapshot_time = C(200.0);

void write(NUMERIC_TYPE* snapshot, const int xsz, NUMERIC_TYPE t)
{
	char filename[256];
	sprintf(filename, "results/%.3f.dat", t);
	FILE* file = fopen(filename, "wb");
	for (int i=0; i<xsz; i++)
	{
		fprintf(file, "%.1f\t", snapshot[i]);
		if (i % 80 == 79) fprintf(file, "\n");
	}
	fclose(file);
}

__global__ void update_flow_variables(NUMERIC_TYPE* U) 
{ 
	int global_i = blockIdx.x*blockDim.x + threadIdx.x;

	for (int i=global_i; i<xsz; i+=blockDim.x*gridDim.x)
	{
		U[i] += C(0.1);
	}
} 

__global__ void simulate(NUMERIC_TYPE* U) 
{ 
	NUMERIC_TYPE dt = C(0.1);
	printf("DEV next_snapshot_time=%f\n", next_snapshot_time);

	while (t < end_time && t < next_snapshot_time)
	{
    	update_flow_variables<<<256, 256>>>(U);
		if (hipDeviceSynchronize() != hipSuccess) return;

		t += dt;

		if (t >= end_time)
		{
			finished = true;
		}
	}

	if (t >= next_snapshot_time)
	{
		next_snapshot_time += C(200.0);
		// TODO
	}
} 

int main(int argc, char *argv[]) 
{ 
	const int xsz_H = 2 << 24; // ~16M elements
	const size_t U_size = xsz_H*sizeof(NUMERIC_TYPE);

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(xsz), &xsz_H, sizeof(int)));

	NUMERIC_TYPE* snapshot_H;
	checkCudaErrors(hipHostMalloc(&snapshot_H, U_size));
	memset(snapshot_H, 0, U_size);

	NUMERIC_TYPE* current_solution_D;
	checkCudaErrors(hipMalloc(&current_solution_D, U_size));
	checkCudaErrors(hipMemcpy(current_solution_D, snapshot_H, U_size, hipMemcpyHostToDevice));

	auto future = std::async(std::launch::async, []{});
	printf("HOST initialised\n");

	while (!finished)
	{
		simulate<<<1,1>>>(current_solution_D); 
		future.wait();
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipMemcpy(snapshot_H, current_solution_D, U_size, hipMemcpyDeviceToHost));
		printf("HOST t=%f\n", t);
		const int t_H = t;
		future = std::async(std::launch::async, [&]{write(snapshot_H, xsz_H, t_H);});
	}

	future.wait();
	checkCudaErrors(hipHostFree(snapshot_H));
	checkCudaErrors(hipFree(current_solution_D));

    return 0; 
}
